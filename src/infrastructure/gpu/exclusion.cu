#include "hip/hip_runtime.h"
#include "soro/infrastructure/gpu/exclusion.h"

#include <cstdio>

namespace soro::infrastructure {

__global__ void hello_world() { printf("Hello World from the GPU!\n"); }

void hello_world_cpu() {
  hello_world<<<1, 1>>>();
  hipDeviceSynchronize();
}

}  // namespace soro::infrastructure